#include "hip/hip_runtime.h"
#include "parrot.hpp"

auto main() -> int {
    auto m = parrot::range(6).as<float>().reshape({2, 3});

    // softmax
    auto cols = m.shape()[1];
    auto z    = m - m.maxr<2>().replicate(cols);
    auto num  = z.exp();
    auto den  = num.sum<2>();
    (num / den.replicate(cols)).print();

    return 0;
}

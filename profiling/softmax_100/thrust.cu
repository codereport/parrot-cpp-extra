#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/transform.h>
#include <cmath>
#include <cub/device/device_segmented_reduce.cuh>
#include <iomanip>
#include <iostream>

// Functor to compute row index from linear index
struct row_index_functor
{
  int cols;
  row_index_functor(int cols) : cols(cols) {}
  __host__ __device__ int operator()(int idx) const { return idx / cols; }
};

// Functor to subtract row max from each element
struct subtract_row_max_functor
{
  const float *data;
  const float *row_maxes;
  int cols;

  subtract_row_max_functor(const float *data,
                           const float *row_maxes,
                           int cols)
      : data(data), row_maxes(row_maxes), cols(cols) {}

  __host__ __device__ float operator()(int idx) const
  {
    int row = idx / cols;
    return data[idx] - row_maxes[row];
  }
};

// Functor to divide by row sum
struct divide_by_row_sum_functor
{
  const float *data;
  const float *row_sums;
  int cols;

  divide_by_row_sum_functor(const float *data,
                            const float *row_sums,
                            int cols)
      : data(data), row_sums(row_sums), cols(cols) {}

  __host__ __device__ float operator()(int idx) const
  {
    int row = idx / cols;
    return data[idx] / row_sums[row];
  }
};

// Helper function to perform segmented reduction (row-wise max)
void segmented_max(const thrust::device_vector<float> &input,
                   thrust::device_vector<float> &output,
                   int rows,
                   int cols)
{
  // Create offset array for segments (0, cols, 2*cols, ...)
  thrust::device_vector<int> offsets(rows + 1);
  thrust::transform(thrust::make_counting_iterator(0),
                    thrust::make_counting_iterator(rows + 1),
                    offsets.begin(),
                    [cols] __device__(int i)
                    { return i * cols; });

  // Perform segmented reduction to find max in each row
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedReduce::Max(
      nullptr,
      temp_storage_bytes,
      thrust::raw_pointer_cast(input.data()),
      thrust::raw_pointer_cast(output.data()),
      rows,
      thrust::raw_pointer_cast(offsets.data()),
      thrust::raw_pointer_cast(offsets.data()) + 1);

  thrust::device_vector<char> temp_storage(temp_storage_bytes,
                                           thrust::default_init);
  hipcub::DeviceSegmentedReduce::Max(
      thrust::raw_pointer_cast(temp_storage.data()),
      temp_storage_bytes,
      thrust::raw_pointer_cast(input.data()),
      thrust::raw_pointer_cast(output.data()),
      rows,
      thrust::raw_pointer_cast(offsets.data()),
      thrust::raw_pointer_cast(offsets.data()) + 1);
}

// Helper function to perform segmented sum (row-wise sum)
void segmented_sum(const thrust::device_vector<float> &input,
                   thrust::device_vector<float> &output,
                   int rows,
                   int cols)
{
  // Create offset array for segments (0, cols, 2*cols, ...)
  thrust::device_vector<int> offsets(rows + 1);
  thrust::transform(thrust::make_counting_iterator(0),
                    thrust::make_counting_iterator(rows + 1),
                    offsets.begin(),
                    [cols] __device__(int i)
                    { return i * cols; });

  // Perform segmented reduction to find sum in each row
  size_t temp_storage_bytes = 0;
  hipcub::DeviceSegmentedReduce::Sum(
      nullptr,
      temp_storage_bytes,
      thrust::raw_pointer_cast(input.data()),
      thrust::raw_pointer_cast(output.data()),
      rows,
      thrust::raw_pointer_cast(offsets.data()),
      thrust::raw_pointer_cast(offsets.data()) + 1);

  thrust::device_vector<char> temp_storage(temp_storage_bytes,
                                           thrust::default_init);
  hipcub::DeviceSegmentedReduce::Sum(
      thrust::raw_pointer_cast(temp_storage.data()),
      temp_storage_bytes,
      thrust::raw_pointer_cast(input.data()),
      thrust::raw_pointer_cast(output.data()),
      rows,
      thrust::raw_pointer_cast(offsets.data()),
      thrust::raw_pointer_cast(offsets.data()) + 1);
}

int main()
{
  const int rows = 100;
  const int cols = 100;
  const int size = rows * cols;

  // Create input matrix: range(10000) -> [0, 1, 2, ..., 9999] reshaped to 100x100
  thrust::host_vector<float> h_input(size);
  for (int i = 0; i < size; ++i)
  {
    h_input[i] = static_cast<float>(i);
  }

  thrust::device_vector<float> d_input = h_input;

  // Step 1: Find row-wise maximum
  thrust::device_vector<float> d_row_max(rows, thrust::default_init);
  segmented_max(d_input, d_row_max, rows, cols);

  // Step 2: Subtract row max from each element (z = m -
  // m.maxr<2>().replicate(cols))
  thrust::device_vector<float> d_z(size, thrust::default_init);
  thrust::transform(
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(size),
      d_z.begin(),
      subtract_row_max_functor(thrust::raw_pointer_cast(d_input.data()),
                               thrust::raw_pointer_cast(d_row_max.data()),
                               cols));

  // Step 3: Apply exponential function (num = z.exp())
  thrust::device_vector<float> d_num(size, thrust::default_init);
  thrust::transform(
      d_z.begin(), d_z.end(), d_num.begin(), [] __device__(float x)
      { return expf(x); });

  // Step 4: Compute row-wise sum of exponentials (den = num.sum<2>())
  thrust::device_vector<float> d_den(rows, thrust::default_init);
  segmented_sum(d_num, d_den, rows, cols);

  // Step 5: Divide each element by its row sum (num / den.replicate(cols))
  thrust::device_vector<float> d_result(size, thrust::default_init);
  thrust::transform(
      thrust::make_counting_iterator(0),
      thrust::make_counting_iterator(size),
      d_result.begin(),
      divide_by_row_sum_functor(thrust::raw_pointer_cast(d_num.data()),
                                thrust::raw_pointer_cast(d_den.data()),
                                cols));

  // Copy result back to host and print
  thrust::host_vector<float> h_result = d_result;

  std::cout << std::fixed << std::setprecision(6);
  for (int i = 0; i < rows; ++i)
  {
    for (int j = 0; j < cols; ++j)
    {
      std::cout << h_result[i * cols + j];
      if (j < cols - 1)
        std::cout << " ";
    }
    std::cout << std::endl;
  }

  return 0;
}
